#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/OpMathType.h>

namespace at { namespace native {

void acos_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "acos_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::acos(a);
        });
      });
}

const char asin_name[] = "asin";
void asin_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if(at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR
  static const auto asin_string = jiterator_stringify(
    template <typename T>
    T asin(T a) {
        return std::asin(a);
    }
  );
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "asin_name", [&]() {
    jitted_gpu_kernel<
        /*name=*/ asin_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, asin_string);
  });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "asin_name", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        return ::asin(static_cast<opmath_t>(a));
    });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, common_dtype, "asin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ::asin(a);
    });
  });
  }
}

const char atan_name[] = "atan";
void atan_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR
  static const auto atan_string = jiterator_stringify(
    template <typename T>
    T atan(T a) {
        return std::atan(a);
    }
  );
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "atan_name", [&]() {
    jitted_gpu_kernel<
        /*name=*/ atan_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, atan_string);
  });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "atan_name", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        return ::atan(static_cast<opmath_t>(a));
    });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      common_dtype, "atan_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::atan(a);
        });
      });
  }
}

const char sin_name[] = "sin";
void sin_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if(at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR
  static const auto sin_string = jiterator_stringify(
    template <typename T>
    T sin(T a) {
        return std::sin(a);
    }
  );
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "sin_name", [&]() {
    jitted_gpu_kernel<
        /*name=*/ sin_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, sin_string);
  });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "sin_name", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        return ::sin(static_cast<opmath_t>(a));
    });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
    ScalarType::Half, ScalarType::BFloat16,
    common_dtype, "sin_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::sin(a);
        });
      });
  }
}

void cos_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "cos_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::cos(a);
        });
      });
}

void sinh_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "sinh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::sinh(a);
        });
      });
}

void cosh_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "cosh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::cosh(a);
        });
      });
}

void tanh_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "tanh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::tanh(a);
        });
      });
}

void acosh_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "acosh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::acosh(a);
        });
      });
}

void asinh_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "asinh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::asinh(a);
        });
      });
}

void atanh_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "atanh_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::atanh(a);
        });
      });
}

const char tan_name[] = "tan";
void tan_kernel_cuda(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (at::isComplexType(common_dtype)) {
#if AT_USE_JITERATOR
  static const auto tan_string = jiterator_stringify(
    template <typename T>
    T tan(T a) {
        return std::tan(a);
    }
  );
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "tan_name", [&]() {
    jitted_gpu_kernel<
        /*name=*/ tan_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, tan_string);
  });
#else
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, common_dtype, "tan_name", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        using opmath_t = at::opmath_type<scalar_t>;
        return ::tan(static_cast<opmath_t>(a));
    });
  });
#endif
  } else {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      common_dtype, "tan_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::tan(a);
        });
      });
  }
}

REGISTER_DISPATCH(acos_stub, &acos_kernel_cuda);
REGISTER_DISPATCH(acosh_stub, &acosh_kernel_cuda);
REGISTER_DISPATCH(asinh_stub, &asinh_kernel_cuda);
REGISTER_DISPATCH(atanh_stub, &atanh_kernel_cuda);
REGISTER_DISPATCH(asin_stub, &asin_kernel_cuda);
REGISTER_DISPATCH(atan_stub, &atan_kernel_cuda);
REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);
REGISTER_DISPATCH(cos_stub, &cos_kernel_cuda);
REGISTER_DISPATCH(sinh_stub, &sinh_kernel_cuda);
REGISTER_DISPATCH(cosh_stub, &cosh_kernel_cuda);
REGISTER_DISPATCH(tanh_stub, &tanh_kernel_cuda);
REGISTER_DISPATCH(tan_stub, &tan_kernel_cuda);

}} // namespace at::native
